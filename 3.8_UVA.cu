#include "common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>


int main(int argc, char*argv[])
{
    int nDeviceNumber = 0;
    int error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("NO Cuda campatable GPU found\n");
        return -1;
    }

    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("Fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }
    
    float *d_mem = NULL;
    ErrorCheck(hipMalloc((void **)&d_mem, sizeof(d_mem)), __FILE__, __LINE__);

    hipPointerAttribute_t pt_Attribute;
    ErrorCheck(hipPointerGetAttributes(&pt_Attribute, d_mem), __FILE__, __LINE__);
    printf("Pionter Attribute:device = %d, devicePointer=%p, type=%d\n",
        pt_Attribute.device, pt_Attribute.devicePointer, pt_Attribute.type
    );
    hipFree(d_mem);
    hipDeviceReset();


    return 0;
}