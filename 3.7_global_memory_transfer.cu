#include "hip/hip_runtime.h"
#include "common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ float factor = 3.2;

__global__ void globalMemory()
{
    printf("Get constant memory:%.2f\n", factor);
    factor += 2;
}

int main(int argc, char*argv[])
{
    int nDeviceNumber = 0;
    int error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("NO Cuda campatable GPU found\n");
        return -1;
    }

    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("Fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }
    dim3 block(1,1);
    dim3 grid(1,1);

    float h_A = 3.6;

    ErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(factor), &h_A, sizeof(float), 0, hipMemcpyHostToDevice), __FILE__, __LINE__);


    globalMemory<<<grid, block>>>();
    hipDeviceSynchronize();

    ErrorCheck(hipMemcpyFromSymbol(&h_A, HIP_SYMBOL(factor), sizeof(float), 0, hipMemcpyDeviceToHost), __FILE__, __LINE__);
    printf("hipMemcpyFromSymbol result is %.2f\n", h_A);

    // get global address
    float *pd_A;
    ErrorCheck(hipGetSymbolAddress((void **)&pd_A, factor), __FILE__, __LINE__);
    hipMemcpy(&h_A, pd_A, sizeof(float), hipMemcpyDeviceToHost);
    printf("hipGetSymbolAddress result is %.2f\n", h_A);

    // reset device
    hipDeviceReset();

    return 0;
}