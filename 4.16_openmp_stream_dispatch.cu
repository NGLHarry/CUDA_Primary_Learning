#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "common/common.h"
#include <stdio.h>
#include <ctime>
#include <omp.h>

#define NSTREAM 4

__device__ void kernel_func()
{
    double sum =0.0;
    long i = 99999;
    while(i > 0)
    {
        for(long j = 0; j < 99999; ++j)
        {
            sum = sum + tan(0.1) * tan(0.1);
        }
        i--;
    }
}

__global__ void kernel_1(int stream)
{
    if(0 == threadIdx.x)
    {
        printf("kernel_1 is excuted in stream %d\n", stream);
    }
    kernel_func();
}


__global__ void kernel_2(int stream)
{
    if(0 == threadIdx.x)
    {
        printf("kernel_2 is excuted in stream %d\n", stream);
    }
    kernel_func();
}

__global__ void kernel_3(int stream)
{
    if(0 == threadIdx.x)
    {
        printf("kernel_3 is excuted in stream %d\n", stream);
    }
    kernel_func();
}
__global__ void kernel_4(int stream)
{
    if(0 == threadIdx.x)
    {
        printf("kernel_4 is excuted in stream %d\n", stream);
    }
    kernel_func();
}

int main(int argc, char** argv)
{
    // get GPU device Count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA capatable GPU found\n");
        return -1;
    }

    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }

    float elapsed_time;
    // Allocate and initialize an array of stream handles
    int n_stream = NSTREAM;
    hipStream_t *streams = (hipStream_t *)malloc(n_stream * sizeof(hipStream_t));

    for(int i = 0; i < n_stream; ++i)
    {
        ErrorCheck(hipStreamCreate(&streams[i]),__FILE__, __LINE__);
    }

    // set up execution configuration
    dim3 block(1);
    dim3 grid(1);

    // create event
    hipEvent_t start, stop;
    ErrorCheck(hipEventCreate(&start, hipEventDefault), __FILE__, __LINE__);
    ErrorCheck(hipEventCreate(&stop, hipEventDefault), __FILE__, __LINE__);

    // record start event
    ErrorCheck(hipEventRecord(start, 0), __FILE__, __LINE__);

    // execute kernels
    // dispatch with OpenMP
    omp_set_num_threads(NSTREAM);
    #pragma omp parallel
    {
        int threadid = omp_get_thread_num();
        kernel_1<<<grid, block, 0, streams[threadid]>>>(threadid);
        kernel_2<<<grid, block, 0, streams[threadid]>>>(threadid);
        kernel_3<<<grid, block, 0, streams[threadid]>>>(threadid);
        kernel_4<<<grid, block, 0, streams[threadid]>>>(threadid);
    }

    // recored stop event
    ErrorCheck(hipEventRecord(stop, 0), __FILE__, __LINE__);
    printf("begin to Synchronize\n");
    ErrorCheck(hipEventSynchronize(stop), __FILE__, __LINE__);

    // calculate elapsed time
    ErrorCheck(hipEventElapsedTime(&elapsed_time, start, stop), __FILE__, __LINE__);
    printf("Mesured time for parallel execution = %.3f ms\n", elapsed_time);

    // realse all stream
    for(int i = 0; i < n_stream; ++i)
    {
        ErrorCheck(hipStreamDestroy(streams[i]), __FILE__, __LINE__);
    }
    free(streams);

    ErrorCheck(hipEventDestroy(start), __FILE__, __LINE__);
    ErrorCheck(hipEventDestroy(stop), __FILE__, __LINE__);


    hipDeviceReset();

    return 0;
}