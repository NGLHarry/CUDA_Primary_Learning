#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "common/common.h"

__global__ void thread_block_fence()
{
   __shared__ float shared;
   shared = 0.0;

   int id = threadIdx.x + blockIdx.x * blockDim.x;
   if((id / 32) == 0 && id == 0)
   {
    shared = 5.0;
   }
   else if((id /32) != 0 && id == 32)
   {
    shared = 7.0;
   }
   __threadfence_block();
   printf("access local shred in thread_barrier,shared=%.2f, blockIdx = %d, threadIdx = %d, threadId = %d\n",
    shared, blockIdx.x, threadIdx.x, id);
}



int main(int argc, char* argv[])
{
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA Campatable GPU found\n");
        return -1;
    }
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {   
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }

    dim3 block(32);
    dim3 grid(2);
    thread_block_fence<<<grid, block>>>();
    hipDeviceSynchronize();
    

    hipDeviceReset();


    return 0;
}