#include "common/common.h"
#include <stdio.h>

int main()
{
    float* gpuMemory = NULL;
    ErrorCheck(hipMalloc(&gpuMemory, sizeof(float)), __FILE__, __LINE__);
    ErrorCheck(hipFree(gpuMemory), __FILE__, __LINE__);
    ErrorCheck(hipFree(gpuMemory), __FILE__, __LINE__);
    ErrorCheck(hipDeviceReset(), __FILE__, __LINE__);
    return 0;
}