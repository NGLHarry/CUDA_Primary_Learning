#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "common/common.h"
#include <stdio.h>
#include <ctime>

// * 设置GPU设备
// * 初始化矩阵
// * 定义CUDA内核
// * 分配GPU内存
// * 将数据传入GPU内存并计算
// * 在主机中获取计算结果

#define NSTREAM 4
#define BDIM 128

void initialData(float* ip, int size)
{
    // generate different seed for random number
    time_t t;
    srand((unsigned int)time(&t));
    for(int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() % 0xFF) / 10.0f;
    } 
    printf("\n");
    return;
}

__global__ void sumArraysOnGPU(float* A, float* B, float* C, const int N)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < N)
    {
        for(int i = 0; i < 9999; ++i)
            C[idx] = A[idx] + B[idx];
    }
}

int main(int argc, char** argv)
{
    // get GPU device Count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA capatable GPU found\n");
        return -1;
    }

    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }

    // set up data size of vectors
    int nElem = 1 << 18;

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);

    // malloc pinned host memory for async memcpy
    float *h_A, *h_B, *gpuRef;
    hipHostAlloc((void **)&h_A, nBytes, hipHostMallocDefault);
    hipHostAlloc((void **)&h_B, nBytes, hipHostMallocDefault);
    hipHostAlloc((void **)&gpuRef, nBytes, hipHostMallocDefault);

    // initialize data at host side
    initialData(h_A, nElem);
    initialData(h_B, nElem);
    memset(gpuRef, 0, nBytes);

    // allocate GPU memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A,nBytes);
    hipMalloc(&d_B,nBytes);
    hipMalloc(&d_C,nBytes);
   
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    // calculate on GPU
    dim3 block(BDIM);
    dim3 grid((nElem + block.x - 1)/ block.x);
    printf("> grid (%d, %d) block (%d, %d)\n", grid.x, grid.y, block.x, block.y);

    // grid parallel operation
    int iElem = nElem / NSTREAM;
    size_t iBytes = iElem * sizeof(float);
    grid.x = (iElem + block.x - 1) / block.x;

    hipStream_t stream[NSTREAM];
    for(int i = 0; i < NSTREAM; ++i)
    {
        hipStreamCreate(&stream[i]);
    }
    hipEventRecord(start, 0);

    for(int i = 0; i < NSTREAM; ++i)
    {
        int ioffset = i * iElem;
        hipMemcpyAsync(&d_A[ioffset], &h_A[ioffset], iBytes, hipMemcpyHostToDevice, stream[i]);
        hipMemcpyAsync(&d_B[ioffset], &h_A[ioffset], iBytes, hipMemcpyHostToDevice, stream[i]);
        sumArraysOnGPU<<<grid, block, 0, stream[i]>>>(&d_A[ioffset], &d_B[ioffset], &d_C[ioffset], iElem);
        hipMemcpyAsync(&gpuRef[ioffset], &d_C[ioffset], iBytes, hipMemcpyDeviceToHost, stream[i]);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float execution_time;
    hipEventElapsedTime(&execution_time, start, stop);

    printf("\n");
    printf("Actual result from overlapped data transfers:\n");
    printf("overlap with %d streams:%f ms (%f GB/s)\n", NSTREAM, execution_time, (nBytes * 2e-6)/ execution_time);


    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(gpuRef);


    hipEventDestroy(start);
    hipEventDestroy(stop);

    for(int i = 0; i < NSTREAM; ++i)
    {
        hipStreamDestroy(stream[i]);
    }
    hipDeviceReset();

    return 0;
}
