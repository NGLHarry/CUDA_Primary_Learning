#include <hip/hip_runtime.h>
#include "common/common.h"
#include <stdio.h>
#include <ctime>



int main(int argc, char** argv)
{
    // get GPU device Count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA capatable GPU found\n");
        return -1;
    }

    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    if(deviceProp.concurrentKernels)
    {
        printf("concurrent kernel is supported on this GPU, conCurrentKernels:%d\n", deviceProp.concurrentKernels);
    }
    else
    {
        printf("concurrent kernel is not supported on this GPU\n");
    }


    hipDeviceReset();

    return 0;
}