#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "common/common.h"
#include <stdio.h>
#include <ctime>


__global__ void infiniteKernel()
{
    while(true)
    {
        
    }
}

int main(int argc, char** argv)
{
    // get GPU device Count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA capatable GPU found\n");
        return -1;
    }

    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }

    // set up data size of vectors
    int nElem = 32;

    // calculate on GPU
    dim3 block(nElem);
    dim3 grid(1);  

    hipStream_t kernel_stream;
    hipStreamCreate(&kernel_stream);
    infiniteKernel<<<grid, block, 0, kernel_stream>>>();

    hipEvent_t kernel_event;
    ErrorCheck(hipEventCreateWithFlags(&kernel_event, hipEventDefault), __FILE__, __LINE__);
    ErrorCheck(hipEventRecord(kernel_event, kernel_stream), __FILE__, __LINE__);



    // wait for data copy to complete
    hipEventSynchronize(kernel_event);
    printf("Event cp_evt is finished\n");
    hipDeviceSynchronize();


    hipStreamDestroy(kernel_stream);
    hipEventDestroy(kernel_event);


    hipDeviceReset();

    return 0;
}