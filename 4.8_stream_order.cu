#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "common/common.h"
#include <stdio.h>
#include <ctime>

// * 设置GPU设备
// * 初始化矩阵
// * 定义CUDA内核
// * 分配GPU内存
// * 将数据传入GPU内存并计算
// * 在主机中获取计算结果

void initialData(float* ip, int size)
{
    // generate different seed for random number
    time_t t;
    srand((unsigned int)time(&t));
    for(int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() % 0xFF) / 10.0f;
    } 
    printf("\n");
    return;
}

__global__ void infiniteKernel()
{
    while(true)
    {
        
    }
}

int main(int argc, char** argv)
{
    // get GPU device Count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA capatable GPU found\n");
        return -1;
    }

    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }

    // set up data size of vectors
    int nElem = 32;

    // malloc host memory
    float *pinned_A;
    size_t nBytes = nElem * sizeof(float);

    ErrorCheck(hipHostAlloc((void **)&pinned_A, nBytes, hipHostMallocDefault),__FILE__, __LINE__);

    // initialize data at host side
    initialData(pinned_A, nElem);

    // allocate GPU memory
    float *d_A;
    hipMalloc((float**)&d_A, nBytes);

    // calculate on GPU
    dim3 block(nElem);
    dim3 grid(2);  
    infiniteKernel<<<grid, block>>>();

    // transfer data from host to device
    hipStream_t data_stream;
    // hipStreamCreate(&data_stream); //blocking stream
    hipStreamCreateWithFlags(&data_stream, hipStreamNonBlocking); //non-blocking stream


    hipMemcpyAsync(d_A, pinned_A, nBytes, hipMemcpyHostToDevice, data_stream);

    hipEvent_t cp_evt;
    ErrorCheck(hipEventCreate(&cp_evt), __FILE__, __LINE__);
    ErrorCheck(hipEventRecord(cp_evt, data_stream), __FILE__, __LINE__);

    // wait for data copy to complete
    hipEventSynchronize(cp_evt);
    printf("Event cp_evt is finished\n");
    hipDeviceSynchronize();

    hipHostFree(pinned_A);

    hipFree(d_A);

    hipStreamDestroy(data_stream);
    hipEventDestroy(cp_evt);


    hipDeviceReset();

    return 0;
}