#include <hip/hip_runtime.h>
#include "common/common.h"
#include <stdio.h>
#include <ctime>

// * 设置GPU设备
// * 初始化矩阵
// * 定义CUDA内核
// * 分配GPU内存
// * 将数据传入GPU内存并计算
// * 在主机中获取计算结果

void initialData(float* ip, int size)
{
    // generate different seed for random number
    time_t t;
    srand((unsigned int)time(&t));
    for(int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() % 0xFF) / 10.0f;
    } 
    printf("\n");
    return;
}


int main(int argc, char** argv)
{
    // get GPU device Count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA capatable GPU found\n");
        return -1;
    }

    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }

    // set up data size of vectors
    int nElem = 1 << 24;

    // malloc host memory
    float *pinned_A;
    size_t nBytes = nElem * sizeof(float);

    ErrorCheck(hipHostAlloc((void **)&pinned_A, nBytes, hipHostMallocDefault),__FILE__, __LINE__);

    // initialize data at host side
    initialData(pinned_A, nElem);

    // allocate GPU memory
    float *d_A;
    hipMalloc((float**)&d_A,nBytes);

    // transfer data from host to device
    hipStream_t data_stream;
    hipStreamCreate(&data_stream);

    hipEvent_t begin_event;
    ErrorCheck(hipEventCreate(&begin_event), __FILE__, __LINE__);
    ErrorCheck(hipEventRecord(begin_event, data_stream), __FILE__, __LINE__);

    hipMemcpyAsync(d_A, pinned_A, nBytes, hipMemcpyHostToDevice, data_stream);

    hipEvent_t end_event;
    ErrorCheck(hipEventCreate(&end_event), __FILE__, __LINE__);
    ErrorCheck(hipEventRecord(end_event, data_stream), __FILE__, __LINE__);

    // wait for data copy to complete
    hipEventSynchronize(end_event);

    float timeElaps = 0.0;
    hipEventElapsedTime(&timeElaps, begin_event, end_event);

    printf("time elapse for data copy from host to device is %.2f(ms)\n",timeElaps);

    hipHostFree(pinned_A);

    hipFree(d_A);

    hipStreamDestroy(data_stream);
    hipEventDestroy(begin_event);
    hipEventDestroy(end_event);


    hipDeviceReset();

    return 0;
}