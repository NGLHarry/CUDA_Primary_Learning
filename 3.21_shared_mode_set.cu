#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "common/common.h"


extern __shared__ int dynamic_array[];

__global__ void dynamic_shared_mem()
{
   dynamic_array[threadIdx.x] = threadIdx.x;
   printf("access dynamic_array in kernel, dynamic_array[%d]=%d\n",threadIdx.x, dynamic_array[threadIdx.x]);
}



int main(int argc, char* argv[])
{
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA Campatable GPU found\n");
        return -1;
    }
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {   
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }

    hipSharedMemConfig sharedMemConfig;
    ErrorCheck(hipDeviceGetSharedMemConfig(&sharedMemConfig), __FILE__, __LINE__);
    printf("current shared memory mode:%d\n",sharedMemConfig);
    if(hipSharedMemBankSizeEightByte != sharedMemConfig)
    {
        sharedMemConfig = hipSharedMemBankSizeEightByte;
        ErrorCheck(hipDeviceSetSharedMemConfig(sharedMemConfig), __FILE__, __LINE__);
    }
    else if(hipSharedMemBankSizeFourByte != sharedMemConfig)
    {
        sharedMemConfig = hipSharedMemBankSizeFourByte;
        ErrorCheck(hipDeviceSetSharedMemConfig(sharedMemConfig), __FILE__, __LINE__);
    }
    printf("current shared memory mode, after modify:%d\n", sharedMemConfig);

    hipDeviceReset();


    return 0;
}