#include "hip/hip_runtime.h"
#include "common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ float factor = 3.2;

__global__ void globalMemory(float *out)
{
    printf("Get constant memory:%.2f\n", factor);
    *out = factor;
}

int main(int argc, char*argv[])
{
    int nDeviceNumber = 0;
    int error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("NO Cuda campatable GPU found\n");
        return -1;
    }

    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("Fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }
    dim3 block(1,1);
    dim3 grid(1,1);

    float *d_A;
    float h_A;

    hipMalloc((void **)&d_A, sizeof(float));

    globalMemory<<<grid, block>>>(d_A);
    hipMemcpy(&h_A, d_A, sizeof(float), hipMemcpyDeviceToHost);
    printf("host memory:%.2f\n", h_A);
    hipDeviceSynchronize();

    // reset device
    hipFree(d_A);
    hipDeviceReset();

    return 0;
}