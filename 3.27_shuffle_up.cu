#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "common/common.h"


__global__ void shfl_up(int *in, int *out, int const srcLane)
{
   int value = in[threadIdx.x];
   value = __shfl_up(value, srcLane, 16);
   out[threadIdx.x] = value;
}



int main(int argc, char* argv[])
{
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNumber == 0)
    {
        printf("No CUDA Campatable GPU found\n");
        return -1;
    }
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {   
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }

    // set up data size of vectors
    int nElem = 32;
    // malloc host memory
    int *in = NULL;
    int *out = NULL;
    ErrorCheck(hipHostAlloc((void **)&in, sizeof(int) * nElem, hipHostMallocDefault), __FILE__, __LINE__);
    ErrorCheck(hipHostAlloc((void **)&out, sizeof(int) * nElem, hipHostMallocDefault), __FILE__, __LINE__);
    for(int i = 0;i < nElem; ++i)
    {
        in[i] = i;
    }

    dim3 block(nElem);
    dim3 grid(1);
    shfl_up<<<grid, block>>>(in, out, 3);

    hipDeviceSynchronize();

    for(int i =0;i < nElem; ++i)
    {
        printf("out element is ,id=%d, value=%d\n",i, out[i]);
    }

    hipHostFree(in);
    hipHostFree(out);

    hipDeviceReset();


    return 0;
}