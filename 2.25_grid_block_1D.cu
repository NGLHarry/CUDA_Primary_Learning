#include "hip/hip_runtime.h"
#include "common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void sumMatrixOnGPU2D(int *MatA, int *MatB, int *MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    if(ix < nx)
    {
        for(int iy = 0; iy < ny; ++iy)
        {
            int idx = iy * nx +ix;
            MatC[idx] = MatA[idx] + MatB[idx];
        }
    }

    
}

int main(int argc, char **argv)
{
    if(argc != 2)
        return -1;
    int block_x = atoi(argv[1]);
    int block_y = 1;

    int nDeviceNum = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNum), __FILE__, __LINE__);
    if(error != hipSuccess || nDeviceNum == 0)
    {
        printf("No CUDA compatable GPU found!\n");
        return -1;
    }

    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing\n");
        return -1;
    }
    else
    {
        printf("set GPU 0 for computing\n");
    }

    // set matrix dimension
    int nx = 1 << 14;
    int ny = 1 << 14;
    int nxy = nx * ny;
    int nBytes = nxy * sizeof(int);

    // malloc host memory
    int *h_A, *h_B, *gpuRef;
    h_A = (int *)malloc(nBytes);
    h_B = (int *)malloc(nBytes);
    gpuRef = (int *)malloc(nBytes);

    for(int i = 0; i < nxy; ++i)
    {
        h_A[i] = i;
        h_B[i] = i+1;
    }

    // malloc device memory
    int *d_MatA, *d_MatB, *d_MatC;
    hipMalloc((void **)&d_MatA, nBytes);
    hipMalloc((void **)&d_MatB, nBytes);
    hipMalloc((void **)&d_MatC, nBytes);

    // copy data to GPU Memeroy
    hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice);

    // set up execution configuration
    // make each row data excuted in each block
    dim3 block(block_x, block_y);
    dim3 grid((nx + block_x -1)/ block_x, 1);
    printf("Thread config: grid:<%d, %d>, block:<%d, %d>\n",
            grid.x, grid.y, block.x, block.y);

    // invoke the kernel
    double dTime_begin = GetCPUSecond();
    sumMatrixOnGPU2D<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    hipDeviceSynchronize();
    double dTime_end = GetCPUSecond();
    printf("Element Size:%d, Matrix add time Elapse:%.5f\n", nxy, dTime_end-dTime_begin);
    hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost);
    for(int i = 0; i < 10; ++i)
    {
        printf("idx = %d, matrix_A:%d, matrix_B:%d, result = %d\n",i+1, h_A[i],h_B[i], gpuRef[i]);
    }

    hipFree(d_MatA);
    hipFree(d_MatB);
    hipFree(d_MatC);
    free(h_A);
    free(h_B);
    free(gpuRef);
    hipDeviceReset();
    return 0;
}

